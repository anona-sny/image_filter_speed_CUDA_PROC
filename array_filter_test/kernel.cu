#include "hip/hip_runtime.h"
#include ""
#include "stdio.h"
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include <opencv2/imgproc.hpp>
#include <iostream> 
#include <sys/stat.h>
#include <string>
#include <fstream>
#include "AnonaImage.h"
#include <math.h> 

using namespace cv;
using namespace std;
using namespace std::chrono;

struct Filter;

AnonaImageData testCPUMatrixFilter(AnonaImageData * data, Filter filter);

struct Filter {
	vector<float> matrix;
	float divider;
	int radius;
};


int main(int argc, char *argv[])
{
	cout << "You have entered " << argc
		<< " arguments:" << "\n";
	for (int i = 0; i < argc; ++i) {
		cout << "Argument cislo. " << i << ": " << argv[i] << "\n";
	}

	std::vector <std::string> sources;
	std::string imgSRC;
	for (int i = 0; i < argc; ++i) {
		if (std::string(argv[i]) == "--image") {
			if (i + 1 < argc) { // Make sure we aren't at the end of argv!
				i++;
				imgSRC = argv[i]; // Increment 'i' so we don't get the argument as the next argv[i].
			} else { // Uh-oh, there was no argument to the destination option.
				std::cerr << "--image option requires one argument." << std::endl;
				return 1;
			}
		} else {
			sources.push_back(argv[i]);
		}
	}
	cout << "Image: " << imgSRC << std::endl;

	AnonaImage image;
	image.readFile(imgSRC);
	if (image.isError()) {
		cerr << image.getError() << endl;
		return 1;
	}
	imshow("ImageDisplay", image.getImage());
	waitKey(0);

	image.imageToArray();

	/*
	Filter filter;
	filter.matrix.reserve(9);
	filter.matrix = {
		0.5,1,2,1,0.5,
		1,7,10,7,1,
		2,10,50,10,2,
		1,7,10,7,1,
		0.5,1,2,1,0.5
	};
	filter.divider = 136;
	filter.radius = 2;*/

	Filter filter;
	filter.matrix.reserve(9);
	filter.matrix = {
		0, 5, 0,
		5, 10, 5,
		0, 5, 0
	};
	filter.divider = 30;
	filter.radius = 1;

	image.setData(testCPUMatrixFilter(image.getData(), filter));
	image.arrayToImage();

	imshow("ImageDisplay", image.getImage());
	waitKey(0);

	image.saveFile("C:/Users/anona/Desktop/out.jpeg");

	return 0;
}

AnonaImageData testCPUMatrixFilter(AnonaImageData * data, Filter filter) {
	auto start = steady_clock::now();
	// c++ need copy for non pointer working
	AnonaImageData dataCopy;
	dataCopy.width = data->width;
	dataCopy.height = data->height;

	int x;
	int y;
	int width = data->width;
	int height = data->height;
	int size = width * height;

	vector<float> filterMatrix = filter.matrix;

	vector<float> redChannel(size, 0);
	vector<float> greenChannel(size, 0);
	vector<float> blueChannel(size, 0);
	vector<float> alphaChannel(size, 0);

	int radius = filter.radius;
	int radius2Plus = radius * 2 + 1;
	int sizeOfFilterArray = (2 * radius + 1)*(2 * radius + 1);
	float output[4] = { 0, 0, 0, 0 };

	int index = 0;
	int index1 = 0;
	int index2 = 0;

	for (int h = 0; h < height; h++) {
		for (int w = 0; w < width; w++) {
			index = h * width + w;
			if (w < radius || h < radius || w > width - radius - 1 || h > height - radius - 1) {
				redChannel[index] = data->redArray[index];
				greenChannel[index] = data->greenArray[index];
				blueChannel[index] = data->blueArray[index];
				alphaChannel[index] = data->alphaArray[index];
			} else {
				output[0] = 0;
				output[1] = 0;
				output[2] = 0;
				output[3] = 0;
				// image cropped by (radius) pixel around border
				for (int i = 0; i < radius2Plus; i++) {
					for (int j = 0; j < radius2Plus; j++) {
						index1 = (h + (i - radius)) * width + (w + (j - radius));
						index2 = i * radius2Plus + j;
						output[0] += data->redArray[index1] * filterMatrix[index2];
						output[1] += data->greenArray[index1] * filterMatrix[index2];
						output[2] += data->blueArray[index1] * filterMatrix[index2];
						output[3] += data->alphaArray[index1] * filterMatrix[index2];
					}
				}
				redChannel[index] = output[0] / filter.divider;
				greenChannel[index] = output[1] / filter.divider;
				blueChannel[index] = output[2] / filter.divider;
				alphaChannel[index] = output[3] / filter.divider;
			}
		}
	}

	dataCopy.redArray = redChannel;
	dataCopy.greenArray = greenChannel;
	dataCopy.blueArray = blueChannel;
	dataCopy.alphaArray = alphaChannel;

	auto end = steady_clock::now();
	nanoseconds a = duration_cast<nanoseconds>(end - start);
	std::cout << "It took me " << a.count() << " nano." << endl;

	return dataCopy;
}